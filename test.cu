#include "hip/hip_runtime.h"
cd /hy-tmp
cat <<EOF > test.cu
#include <iostream>

__global__ void kernel() {
}

int main() {
    kernel<<<1,1>>>();
    hipDeviceSynchronize();
    if (hipGetLastError() != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl;
        return -1;
    }
    std::cout << "Minimal CUDA C++ program compiled and ran successfully!" << std::endl;
    return 0;
}
EOF